
#include <hip/hip_runtime.h>


// Test if CUDA will propagate compile-time constants
// through function arguments.
__global__ void f(const int n){
	float array[n];
}
int main(int argc, char **argv) 
{
	dim3 gridSize(1,1,1);
	dim3 blockSize(1,1);
	const int n=3;
	f<<<gridSize,blockSize>>>(n);
}
/*
nvcc foo.cu -o foo -lcuda -I../util -I../../util -I../../../util -I../../../util -arch=sm_20 -DBLAS_IMPLEMENTATION_MKL -I/usr/local/intel/mkl//include -L/usr/local/intel/mkl//lib/em64t -lmkl_intel_lp64 -lmkl_intel_thread -lmkl_core -liomp5 -lpthread --profile -Xptxas -v
foo.cu(6): error: constant value is not known

1 error detected in the compilation of "/tmp/tmpxft_00000915_00000000-4_foo.cpp1.ii".
make: *** [foo] Error 2
*/
